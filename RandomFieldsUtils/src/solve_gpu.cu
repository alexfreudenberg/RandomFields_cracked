#include "hip/hip_runtime.h"
#include <hipsolver.h>
#include <hipsolver.h>
#include <hip/hip_runtime_api.h>
#include <hipblaslt.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <vector>

#include "errors_messages.h"
#include "RandomFieldsUtils.h"
#include "solve_gpu.h"
#include "options.h"




__global__ void logdet_kernel(double *d_matrix, Uint *d_size, double *d_logdet){
    __shared__ double logdet_loc;
    __shared__ double submatrix[THREADS_PER_BLOCK];
    logdet_loc = 0.0;
    *d_logdet = 0.0;
    int idx = blockDim.x * blockIdx.x + threadIdx.x,
        thread = threadIdx.x;
    if(idx < *d_size){
        submatrix[thread] = d_matrix[idx * (*d_size +1)];
    }

    __syncthreads();
    atomicAdd(&logdet_loc, idx >= *d_size ? 0 : (log(submatrix[thread])));

    __syncthreads();
    if(threadIdx.x ==0){atomicAdd(d_logdet, logdet_loc);
    };
};

int cholGPU(bool copy, double *matrix, Uint size, double *B, Uint rhs_cols,
     double *LogDet, double *RESULT){
    /*
        This function solves the problem
            A x = b
        on   an available GPU and writes the solution to the original memory
        Input: 
            matrix: pointer to rowwise allocated matrix A
            individuals: number of individuals in matrix, i.e. dimension
            vector: pointer to vector b
        Ouput:
            vector: contains solution x after the function has been called
    */

    //declare/define process variables
    int bufferSize = 0;
    int *info = NULL;
    int h_info = 0;
    double *buffer = NULL;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    hipsolverHandle_t handle = NULL;
    hipStream_t stream = NULL;

    //declare device variables
    double *d_matrix = NULL;
    double *d_B = NULL;
    double *d_logdet = NULL;
    Uint *d_size = NULL;

    //initialize handle and stream, calculate buffer size needed for cholesky
    hipsolverDnCreate(&handle);
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipsolverSetStream(handle, stream);

    hipsolverDnDpotrf_bufferSize(handle, uplo, size, matrix,
        size, &bufferSize);
    //PRINTF("Buffersize: %f", ((float) bufferSize)/1073741824.0);
    hipMalloc(&info, sizeof(int));
    hipMalloc(&buffer, sizeof(double) * bufferSize);
    //allocate memory on device  
    hipMalloc((void**)&d_matrix, sizeof(double) * size * size);
    hipMalloc((void **)&d_B, sizeof(double) * size * rhs_cols);
    hipMemset(info, 0, sizeof(int));

    //copy data to device
    hipMemcpy(d_matrix, matrix, sizeof(double) * size * size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeof(double) * size * rhs_cols, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    //write cholesky factorization to device copy of A
    hipsolverDnDpotrf(handle, uplo, size,
            d_matrix, size, buffer, bufferSize, info);
            
    //Synchronize is necessary, otherwise error code "info" returns nonsense 
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) PRINTF("%s\n", hipGetErrorString(err));

    //check for errors
    hipMemcpy(&h_info, info, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    if (0 != h_info) {
        PRINTF("Error: Cholesky factorization failed\n");
    }
    //calculate x = A\b
    hipsolverDnDpotrs(handle, uplo, size, rhs_cols, 
            d_matrix, size, d_B,
             size, info);

    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) PRINTF("Potrs: %s\n", hipGetErrorString(err));
    
    if(LogDet != NULL){
        hipMalloc((void**)&d_logdet, sizeof(double));
        hipMalloc((void**)&d_size, sizeof(Uint));
        hipMemcpy(d_size, &size, sizeof(Uint), hipMemcpyHostToDevice);
        logdet_kernel <<< (size - 1)/THREADS_PER_BLOCK +1 ,THREADS_PER_BLOCK>>> (d_matrix, d_size, d_logdet);
        hipDeviceSynchronize();
        hipMemcpy(LogDet, d_logdet, sizeof(double), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        hipFree(d_size);
        hipFree(d_logdet);
    }
    err = hipGetLastError();
    if (err != hipSuccess) PRINTF("Err at Logdet: %s\n", hipGetErrorString(err));

     //*LogDet = 1.0;
    //copy  solution from device to vector on host
    hipMemcpy(RESULT, d_B, sizeof(double) * size * rhs_cols, hipMemcpyDeviceToHost);
    err = hipGetLastError();
    if (err != hipSuccess) PRINTF("Memcpy: %s\n", hipGetErrorString(err));
    //free allocated memory
    hipFree(info);
    hipFree(buffer);
    hipFree(d_matrix);
    hipFree(d_B);
    hipsolverDnDestroy(handle);
    hipStreamDestroy(stream);
    return 0;
};



void mgpuSolve(double *matrix, Uint individuals, double *vector){
    /*
        This function solves the problem
            A x = b
        on an MULTIPLE GPUs and writes the solution to the original memory of b
        Input: 
            matrix: pointer to rowwise allocated matrix A
            individuals: number of individuals in matrix, i.e. dimension
            vector: pointer to vector b
        Ouput:
            vector: contains solution x after the function has been called
    */

    // Define auxiliary variables
    cusolverMgHandle_t handle = NULL;
    const int max_devices = 8; // Maximum number of devices to be used
    int nbGpus = 0;
    std::vector<int> deviceList;
    const int N = individuals, lda = N; // Dimension of matrix
    const int IA  = 1;
    const int JA  = 1;
    const int T_A = 256; //Tile size
    const int IB  = 1;
    const int JB  = 1;
    const int T_B = 1000, ldb = N; 
    int info = 0;
    int64_t lwork_potrf = 0, lwork_potrs = 0, lwork = 0 ;

    cudaLibMgMatrixDesc_t descrA, descrB;
    cudaLibMgGrid_t grid;
    double **array_d_A = NULL;
    double **array_d_B = NULL;
    double **array_d_work = NULL;

    // Create handles and select devices
    hipsolverStatus_t status = cusolverMgCreate(&handle);
    if(HIPSOLVER_STATUS_SUCCESS != status)PRINTF("Handle couldn't be created");
    
    hipError_t cudaStat = hipGetDeviceCount( &nbGpus );
    nbGpus = (nbGpus < max_devices)? nbGpus : max_devices;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int cc_major = prop.major, cc_minor = prop.minor;
    for(int i = 0; i< nbGpus; i++){
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        if(prop.major == cc_major & prop.minor == cc_minor)
                deviceList.push_back(i);
    }
    nbGpus = deviceList.size();
    status = cusolverMgDeviceSelect(
        handle,
        nbGpus,
        &deviceList[0]);
    if(HIPSOLVER_STATUS_SUCCESS != status) PRINTF("Devices couldn't be selected.");

    // Enable peer access for selected devices
    for(int i = 0; i < nbGpus; i++){
        hipSetDevice(deviceList[i]);
        for(int j = 0; j< nbGpus; j++){
            if(i==j)continue;
            cudaStat = hipDeviceEnablePeerAccess(deviceList[j],0);
            if(cudaStat != hipSuccess)PRINTF("Device %d can't access device %d.",deviceList[i],deviceList[j]);
            PRINTF("Access enabled for devices (%d,%d)",deviceList[i],deviceList[j]);
        }
    }
    // Create device grid for vectors A, B
    status = cusolverMgCreateDeviceGrid(&grid, 1, nbGpus, &deviceList[0], CUDALIBMG_GRID_MAPPING_COL_MAJOR );
    if(HIPSOLVER_STATUS_SUCCESS != status)PRINTF("Grid couldn't be created.");

    // Creeate matrix descriptions
    status = cusolverMgCreateMatrixDesc(
        &descrA,
        N,   /* nubmer of rows of (global) A */
        N,   /* number of columns of (global) A */
        N,   /* number or rows in a tile */
        T_A, /* number of columns in a tile */
        HIP_R_64F,
        grid );
    if(HIPSOLVER_STATUS_SUCCESS != status)PRINTF("Matrix descriptions couldn't be created.");
    status = cusolverMgCreateMatrixDesc(
        &descrB,
        N,    /* nubmer of rows of (global) B */
        1, /* number of columns of (global) B */
        N,    /* number or rows in a tile */
        T_B,  /* number of columns in a tile */
        HIP_R_64F,
        grid );
    if(HIPSOLVER_STATUS_SUCCESS != status)PRINTF("Matrix description B couldn't be created.");


    // Allocate arrays of device pointers which point at the memory allocated on each device
    array_d_A = (double**) malloc (sizeof(double*) * nbGpus );
    array_d_B = (double**)malloc(sizeof(double*)*nbGpus);
    array_d_work = (double**)malloc(sizeof(double*)*nbGpus);
    memset(array_d_work, 0, sizeof(void*)*nbGpus);

    // Calculate block size on device
    const int A_num_blks = ( N + T_A - 1) / T_A;
    const int B_num_blks = ( N + T_B - 1) / T_B;
    const int A_blks_per_device = (A_num_blks + nbGpus-1)/nbGpus;
    const int B_blks_per_device = (B_num_blks + nbGpus-1)/nbGpus;

    // Allocate memory on each device
    for( int p = 0 ; p < nbGpus ; p++){
        hipSetDevice(deviceList[p]);
        cudaStat = hipMalloc( &(array_d_A[p]), sizeof(double)*lda*T_A*A_blks_per_device );
        if(hipSuccess != cudaStat)PRINTF("Memory for matrix A couldn't be allocated on device %d.",deviceList[p]);
        cudaStat = hipMalloc( &(array_d_B[p]), sizeof(double)*ldb*T_B*B_blks_per_device );
        if(hipSuccess != cudaStat)PRINTF("Memory for matrix B couldn't be allocated on device %d.",deviceList[p]);
    }

    // Copy arrays A and B to device
    for( int k = 0 ; k < A_num_blks ; k++){
    /* k = ibx * nbGpus + p */
        const int p   = (k % nbGpus);
        const int ibx = (k / nbGpus);
        double *h_Ak = matrix + (size_t)lda*T_A*k;
        double *d_Ak = array_d_A[p] + (size_t)lda*T_A*ibx;
        const int width = MIN( T_A, (N - T_A*k) );
        cudaStat = hipMemcpy(d_Ak, h_Ak, sizeof(double)*lda*width, hipMemcpyHostToDevice);
        if(hipSuccess != cudaStat)PRINTF("Matrix A couldn't be copied at block (%d, %d).", p,ibx);
    }
    for( int k = 0 ; k < B_num_blks ; k++){
    /* k = ibx * nbGpus + p */
        const int p   = (k % nbGpus);
        const int ibx = (k / nbGpus);
        double *h_Bk = vector + (size_t) T_B*k;
        double *d_Bk = array_d_B[p] + (size_t) T_B*ibx;
        cudaStat = hipMemcpy(d_Bk, h_Bk, sizeof(double)*T_B, hipMemcpyHostToDevice);
        if(hipSuccess != cudaStat)PRINTF("Matrix B couldn't be copied at block (%d, %d).", p,ibx);
    }

    // Calculate buffersizes necessary for potrf and potrs
    hipDeviceSynchronize();
    status = cusolverMgPotrf_bufferSize(
        handle,
		HIPBLAS_FILL_MODE_LOWER,
        N,
        (void**)array_d_A,
        IA, /* base-1 */
        JA, /* base-1 */
        descrA,
        HIP_R_64F,
        &lwork_potrf);
    if(HIPSOLVER_STATUS_SUCCESS != status)PRINTF("Buffer size potrf couldn't  be calculated");    
    hipDeviceSynchronize();
    status = cusolverMgPotrs_bufferSize(
        handle,
		HIPBLAS_FILL_MODE_LOWER,
        N,
        1, /* number of columns of B */
        (void**)array_d_A,
        IA,
        JA,
        descrA,
        (void**)array_d_B,
        IB,
        JB,
        descrB,
        HIP_R_64F,
        &lwork_potrs);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) PRINTF("Buffersize calculation: %s\n", hipGetErrorString(err));
    if(HIPSOLVER_STATUS_SUCCESS != status)PRINTF("Buffer size potrs couldn't  be calculated");    

    lwork = (lwork_potrf > lwork_potrs)? lwork_potrf : lwork_potrs;

    // Allocate workspace size
    for(int idx = 0 ; idx < nbGpus ; idx++){
        int deviceId = deviceList[idx];
        hipSetDevice( deviceId );
        void *d_workspace = NULL;
        cudaStat = hipMalloc(&d_workspace, sizeof(double)*lwork);
        if( hipSuccess != cudaStat )PRINTF("Workspace couldn't be allocated.");
        ((void**)array_d_work )[idx] = d_workspace;
    }

    // Calculate potrf to workspace
    status = cusolverMgPotrf(
        handle,
		HIPBLAS_FILL_MODE_LOWER,
        N,   
        (void**)array_d_A,
        IA,
        JA,
        descrA,
        HIP_R_64F,
        (void**)array_d_work,
        lwork,
        &info  /* host */
    );
    hipDeviceSynchronize();
    if(HIPSOLVER_STATUS_SUCCESS != status) PRINTF("Potrf couldn't be calculated");
    if(info != 0)PRINTF("Info code %d", info);
    // Calculate potrs to B
    status = cusolverMgPotrs(
        handle,
		HIPBLAS_FILL_MODE_LOWER,
        N,
        1, /* number of columns of B */
        (void**)array_d_A,
        IA,
        JA,
        descrA,
        (void**)array_d_B,
        IB,
        JB,
        descrB,
        HIP_R_64F,
        (void**)array_d_work,
        lwork,
        &info  /* host */
    );
    hipDeviceSynchronize();
    if(HIPSOLVER_STATUS_SUCCESS != status) PRINTF("Potrs couldn't be calculated");
    if(info != 0)PRINTF("Info code %d", info);

    // Copy solution B back to host
    for( int k = 0 ; k < B_num_blks ; k++){
    /* k = ibx * nbGpus + p */
        const int p   = (k % nbGpus);
        const int ibx = (k / nbGpus);
        double *h_Bk = vector + (size_t) T_B*k;
        double *d_Bk = array_d_B[p] + (size_t) T_B*ibx;
        cudaStat = hipMemcpy(h_Bk, d_Bk, sizeof(double)*T_B, hipMemcpyDeviceToHost);
        if(hipSuccess != cudaStat)PRINTF("Matrix B couldn't be copied at block (%d, %d).", p,ibx);
    }

    // Free memory on device and host
    for(int i = 0; i< nbGpus; i++){
        hipSetDevice(deviceList[i]);
        hipDeviceReset();
    }
    free(array_d_A); free(array_d_B); free(array_d_work);
}

void gpu_relmat_cublas(Uint* M, double* A, Uint snps, Uint individuals){
    /*
        Calculates the crossproduct of M with cublas and stores the result in A.
        Input:
            M: non-encoded matrix of dimension snps x indiv (k x n) storing genomic information
            A: pointer to result matrix
            snps: Number of snps
            individuals: number of individuals
        Output:
            A: matrix containing the type-casted result of M^T * M
        
        Note: cublas is fortran based and therefore assumes M is column-major. Therefore to calculate
            A we instruct cublasgemmex to calculate M * M^T and adjust its parameters.
            Furthermore, gemmex requires the matrix M to have a row number that is a multiple of four
            Therefore this function implements a zero-padding to add extra rows
    */
    
    //Define auxiliary variables as needed for gemmex
        Uint n = individuals;
        Uint m = individuals;
        Uint k = snps;
    
    //Auxiliary padding variables for padding
        Uint k_pad_diff = (PADDIM - k % PADDIM) % PADDIM;
        Uint k_pad = k + k_pad_diff;
        Uint dim = m * k_pad;
    
    //Start timing copy and calculation time
    #ifdef DEBUG
        std::chrono::time_point<std::chrono::high_resolution_clock> timer_start;
        std::chrono::time_point<std::chrono::high_resolution_clock> timer_stop;
        timer_start = std::chrono::high_resolution_clock::now();
    #endif
    //Declare cublas variables and allocate memory
        hipblasHandle_t handle;
        hipblasCreate(&handle);
        int8_t *d_M, *h_M;
        int32_t *d_C, *h_C;
        int32_t alpha = 1.f;
        int32_t beta = 0.f;
        hipMalloc(&d_M, sizeof(int8_t) * dim);
        hipMalloc(&d_C, sizeof(int32_t) * n * m );
        hipHostMalloc((void **)&h_M, sizeof(int8_t) * dim);
        hipHostMalloc((void **)&h_C, sizeof(int32_t) * n * m);
    
    
    
    //Type-cast matrix M to int8 and store the result in page-locked memory
    //Zero-pad matrix to get a row number that is a multiple of four
    #ifdef DO_PARALLEL
    #pragma omp parallel for num_threads(CORES)   
    #endif
        for(int i = 0; i < n; i++){
            for(int j = 0; j < k_pad; j++){
            h_M[j + i * k_pad] = (int8_t) (j< k ?  M[j + i * k] : 0 );
            }
        }
    
    
    //Copy int8 matrix to device
    hipMemcpy(d_M, h_M, sizeof(int8_t) * dim, hipMemcpyHostToDevice);  

    //Calculate the crossproduct and check for errros
        hipblasStatus_t stat = hipblasGemmEx(handle,
            HIPBLAS_OP_T,
            HIPBLAS_OP_N,
            n,
            m,
            k_pad,
            &alpha,
            d_M,
            HIP_R_8I,
            k_pad, // I have no idea why this doesnt need to be individuals, same below
            d_M,
            HIP_R_8I,
            k_pad,
            &beta,
            d_C,
            HIP_R_32I,
            n,
            HIP_R_32I, //HIPBLAS_COMPUTE_32I,
            HIPBLAS_GEMM_DEFAULT
            );
        
        if(stat) PRINTF("GemmEx failed.");
        hipDeviceSynchronize();
    
    
    //copy result back to host
        hipMemcpy(h_C, d_C, sizeof(int32_t) * n * m, hipMemcpyDeviceToHost);
    
    //Convert result to double and store it in output matrix A
    #ifdef DO_PARALLEL
    #pragma omp parallel for num_threads(CORES)   
    #endif
        for (int i = 0; i < n * m; i++) A[i] = (double) h_C[i];
    
    //Free memory 
        hipblasDestroy(handle);
        hipFree(d_M);
        hipFree(d_C);
        hipHostFree(h_C);
        hipHostFree(h_M);
    
    //Stop timer
    #ifdef DEBUG
        timer_stop = std::chrono::high_resolution_clock::now();
        PRINTF("Time: %.3f s\n", ((float) std::chrono::duration_cast<std::chrono::microseconds>(timer_stop - timer_start).count())/1000000.0 );
    #endif
    } 
